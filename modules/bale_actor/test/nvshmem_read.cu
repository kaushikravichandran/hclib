#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <exstack.h>
extern "C" {
#include "spmat.h"
}

#define NV_THREADS nvshmem_n_pes()
#define NV_MYTHREAD nvshmem_my_pe()

void nvshmem_write_upc_array_int64(__shared__ int64_t *addr, size_t index, size_t blocksize, int64_t val) {
  int pe;
  size_t local_index;
  int64_t *local_ptr;


  pe = index % nvshmem_n_pes();
  local_index = (index / nvshmem_n_pes())*blocksize;

  local_ptr =(int64_t*)(( (char*)addr ) + local_index);

  nvshmem_int_p ( local_ptr, val, pe );
}

/*! \brief Read a sparse matrix in matrix market format on one PE and create a distributed matrix
  from that.
  * Only PE 0 reads the matrix file.
  * 
  * \param name The name of the file.
  * \return The sparsemat_t struct.
  * \ingroup spmatgrp
  */
sparsemat_t * read_matrix_mm_to_dist_nv(char * name) {
  typedef struct pkg_rowcol_t{
    int64_t row;    
    int64_t col;
  }pkg_rowcol_t;

  int64_t nr, nc, nnz = 0, i, pe;
  __shared__ int64_t * sh_data;
  sh_data = nvshmem_calloc (THREADS*4, sizeof(int64_t));

  int64_t * rowcount;
  edge_t * edges;
  w_edge_t * tri;
  if(!MYTHREAD){
    int fscanfret;
    int64_t * nnz_per_th = calloc(THREADS, sizeof(int64_t));
    
    FILE * fp = fopen(name, "r");
    if( fp == NULL ) {
      fprintf(stderr,"read_matrix_mm: can't open file %s \n", name);
      nvshmem_global_exit(1);
    }
    
    // Read the header line of the MatrixMarket format 
    char * object = calloc(64, sizeof(char));
    char * format = calloc(64, sizeof(char));
    char * field = calloc(64, sizeof(char));;
    fscanfret = fscanf(fp,"%%%%MatrixMarket %s %s %s\n", object, format, field);
    if( (fscanfret != 3 ) || strncmp(object,"matrix",24) || strncmp(format,"coordinate",24) ){
      fprintf(stderr,"read_matrix_mm: Incompatible matrix market format.\n");
      fprintf(stderr,"                First line should be either:\n");
      fprintf(stderr,"                matrix coordinate pattern\n");
      fprintf(stderr,"                OR\n");
      fprintf(stderr,"                matrix coordinate real\n");
      fprintf(stderr,"                OR\n");
      fprintf(stderr,"                matrix coordinate integer\n");
      nvshmem_global_exit(1);
    }

    // Make sure that this is a format we support
    if(strncmp(field,"pattern",24) && strncmp(field,"real",24) && strncmp(field,"integer",24) ){
      fprintf(stderr,"read_matrix_mm: Incompatible matrix market field.\n");
      fprintf(stderr,"                Last entry on first line should be pattern, real, or integer\n");
      nvshmem_global_exit(1);
    }
    int64_t values;
    if(strncmp(field,"pattern",7) == 0){
      values = 0L; // no values
    }else if(strncmp(field,"real",4) == 0){
      values = 1L; // real values
    }else{
      values = 2L; // integer values
    }
    
    // Read the header (nr, nc, nnz)
    fscanfret = fscanf(fp,"%"PRId64" %"PRId64" %"PRId64"\n", &nr, &nc, &nnz);
    if( (fscanfret != 3 ) || (nr<=0) || (nc<=0) || (nnz<=0) ) {
      fprintf(stderr,"read_matrix_mm: reading nr, nc, nnz\n");
      nvshmem_global_exit(1);
    }

    // allocate space to store the matrix data    
    rowcount = calloc(nr, sizeof(int64_t));
    if(!rowcount){
      T0_printf("ERROR: read_matrix_mm_to_dist: could not allocate arrays\n");
      for(i = 0; i < THREADS; i++) nvshmem_write_upc_array_int64(sh_data, i, sizeof(int64_t), -1);
    }
    
    // read the data
    int64_t row, col, val, pos = 0;
    if(values == 0){
      edges = calloc(nnz, sizeof(edge_t));
      while(fscanf(fp,"%"PRId64" %"PRId64"\n", &row, &col) != EOF){
        row--;//MM format is 1-up
        col--;
        edges[pos].row   = row;
        edges[pos++].col = col;
        nnz_per_th[row % THREADS]++;
        rowcount[row]++;
      }
      qsort( edges, nnz, sizeof(edge_t), edge_comp);
    }else{
      tri = calloc(nnz, sizeof(w_edge_t));    
      while(fscanf(fp,"%"PRId64" %"PRId64" %"PRId64"\n", &row, &col, &val) != EOF){
        tri[pos].row = row - 1;
        tri[pos].col = col - 1;
        tri[pos++].val = val;
        nnz_per_th[row % THREADS]++;
        rowcount[row]++;
      }
      qsort( tri, nnz, sizeof(w_edge_t), w_edge_comp);
    }
    
    fclose(fp);
    if(nnz != pos){
      T0_printf("ERROR: read_matrix_mm_to_dist: nnz (%"PRId64") != pos (%"PRId64")\n", nnz, pos);
      for(i = 0; i < THREADS; i++) nvshmem_write_upc_array_int64(sh_data, i, sizeof(int64_t), -1);
    }
    for(i = 0; i < THREADS; i++){
      nvshmem_write_upc_array_int64(sh_data, i, sizeof(int64_t), nnz_per_th[i]);
      nvshmem_write_upc_array_int64(sh_data, i+THREADS, sizeof(int64_t), nr);
      nvshmem_write_upc_array_int64(sh_data, i+2*THREADS, sizeof(int64_t), nc);
      nvshmem_write_upc_array_int64(sh_data, i+3*THREADS, sizeof(int64_t), values);
    }
    free(nnz_per_th);

  }
  
  nvshmem_barrier();

  int64_t * lsh_data = (( int64_t * )((sh_data)+MYTHREAD));
  
  if(lsh_data[0] == -1)
    return(NULL);
  
  int64_t lnnz = lsh_data[0];
  nr = lsh_data[1];
  nc = lsh_data[2];
  int value = (lsh_data[3] != 0L);
  
  sparsemat_t * A = init_matrix(nr, nc, lnnz, value);
  __shared__ int64_t * tmp_offset = nvshmem_calloc(nr + THREADS, sizeof(int64_t));
  if(!A || !tmp_offset){
    T0_printf("ERROR: read_matrix_mm_to_dist: failed to init matrix or tmp_offset!\n");
    return(NULL);
  }

  /* set up offset array and tmp_offset */
  nvshmem_barrier();
  nvshmem_free(sh_data);
  
  if(!MYTHREAD){
    for(i = 0; i < nr; i++)
      nvshmem_write_upc_array_int64(tmp_offset, i, sizeof(int64_t), rowcount[i]);
    free(rowcount);
  }

  nvshmem_barrier();

  int64_t * ltmp_offset = ( int64_t * )((tmp_offset)+MYTHREAD);
  A->loffset[0] = 0;
  for(i = 1; i <= A->lnumrows; i++){
    A->loffset[i] = A->loffset[i-1] + ltmp_offset[i-1];
    ltmp_offset[i-1] = 0;
  }

  int64_t fromth;
  w_edge_t pkg;
  exstack_t * ex = exstack_init(256, sizeof(w_edge_t));
  if( ex == NULL ) return(NULL);
  
  /* distribute the matrix to all other PEs */
  /* pass around the nonzeros */
  /* this is a strange exstack loop since only PE0 has data to push */
  i = 0;
  while(exstack_proceed(ex, (i == nnz))){
    while(i < nnz){
      if(value == 0){
        pkg.row = edges[i].row;
        pkg.col = edges[i].col;
      }else{
        pkg.row = tri[i].row;
        pkg.col = tri[i].col;
        pkg.val = tri[i].val;
      }
      pe = pkg.row % THREADS;
      if(!exstack_push(ex, &pkg, pe))
        break;
      i++;
    }
    exstack_exchange(ex);

    while(exstack_pop(ex, &pkg, &fromth)){
      int64_t row = pkg.row/THREADS;
      int64_t pos = A->loffset[row] + ltmp_offset[row];
      //printf("pos = %ld row = %ld col = %ld\n", pos, row, pkg.col);fflush(0);
      A->lnonzero[pos] = pkg.col;
      if(value) A->lvalue[pos] = pkg.val;
      ltmp_offset[row]++;
    }
  }

  nvshmem_barrier();
  if(!MYTHREAD){
    if(value == 0)
      free(edges);
    else
      free(tri);
  }

  nvshmem_free(tmp_offset);
  exstack_clear(ex);
  sort_nonzeros(A);
  return(A);
}
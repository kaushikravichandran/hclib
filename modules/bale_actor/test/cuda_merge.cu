#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <thrust/set_operations.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/count.h>

__global__ void merge(uint64_t *a, int64_t *b, uint64_t size1, uint64_t size2, uint64_t *counter)
{
    // printf("In kernel: %d %d", size1, size2);
    int i = (int) size1 / 8 * threadIdx.x;
    int j = (int) size2 / 8 * threadIdx.x;
    int endI = (int) size1 / 8 * (threadIdx.x + 1);
    int endJ = (int) size2 / 8 * (threadIdx.x + 1);
    uint64_t lastNum = 0;

    // Merge the two lists
    while (i < endI || j < endJ)
    {
        // printf("Here: %d %d", a[i], b[j]);
        if (a[i] == lastNum || b[j] == lastNum)
        {
            if (a[i] == lastNum)
            {
                i++;
            }
            else
            {
                j++;
            }

            continue;
        }

        (*counter)++;
        if (a[i] < b[j])
        {
            lastNum = a[i];
            i++;
        }
        else
        {
            lastNum = b[j];
            j++;
        }
    }

    // Copy the remaining elements from the first list
    while (i < endI) {
        (*counter)++;
        i++;
    }

    // Copy the remaining elements from the second list
    while (j < endJ) {
        (*counter)++;
        j++;
    }
}

// #define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
// inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
// {
//    if (code != hipSuccess) 
//    {
//       fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
//       if (abort) exit(code);
//    }
// }

uint64_t cuda_merge (uint64_t *a, uint64_t *b, int64_t* output, uint64_t size1, uint64_t size2, uint64_t size3)
{
    // if (size1 == 0 || size2 == 0)
    // {
    //     return 0;
    // }
    // uint64_t size3 = (size1 < size2) ? size1 : size2;
    // int64_t *output;// = (int64_t*)malloc(sizeof(int64_t) * size3);
    // gpuErrchk(hipMalloc((void**)&output, size3 * sizeof(int64_t)));
    // gpuErrchk(hipMemset(output, -1, sizeof(int64_t) * size3));

    thrust::device_ptr<uint64_t> a_start(a);
    thrust::device_ptr<uint64_t> a_end(a + size1);
    thrust::device_ptr<uint64_t> b_start(b);
    thrust::device_ptr<uint64_t> b_end(b + size2);
    thrust::device_ptr<int64_t> output_start(output);

    // printf("In cuda_merge (before): %d %d %d %d %d %d\n", size1, size2, size3, sizeof(a), sizeof(b), sizeof(output));
    thrust::device_ptr<int64_t> output_end = thrust::set_intersection(a_start, a_end, b_start, b_end, output_start, thrust::less<int>());
    //thrust::device_ptr<int64_t> output_end = thrust::set_intersection(thrust::host, a, a + size1, b, b + size2, output_start, thrust::less<int>());
    // hipMemcpy(output_host, output, sizeof(int64_t) * size3, hipMemcpyDeviceToHost);
    // printf("In cuda_merge (after): %d %d %d %d %d %d\n", size1, size2, size3, sizeof(a), sizeof(b), sizeof(output));
    // thrust::device_vector<int64_t> output_vec(output_start, output_start + size3);    
    // int count = thrust::count(output_vec.begin(), output_vec.end(), -1);
    // return size3 - count;
    return (output_end - output_start);
    // printf("In kernel func: %d %d\n", size1, size2);
    // merge<<<1, 8>>>(a, b, size1, size2, counter);
}
